
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <hiprand/hiprand.h>



#define TILE_SIZE 8

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *******************************************************************/
	__shared__ float ds_A[TILE_SIZE][TILE_SIZE];
	__shared__ float ds_B[TILE_SIZE][TILE_SIZE];
	int bx = blockIdx.x;
	int by = blockIdx.y;
        int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = by * blockDim.y + ty;
	int col = bx * blockDim.x + tx;
	float pvalue = 0;
	for ( int p = 0; p < (k - 1)/TILE_SIZE+1; p++){
		if((p*TILE_SIZE + tx < k) && (row < m)){
			ds_A[ty][tx] = A[row * k + p * TILE_SIZE+tx];
		}
		else ds_A[ty][tx] = 0;
		
		if((p*TILE_SIZE + ty < k) && (col < n)){
			ds_B[ty][tx] = B[(p*TILE_SIZE+ty)*n + col];
		}
		else ds_B[ty][tx] = 0;
		__syncthreads();

		if(row < m && col < n){
			for (int i = 0; i <  TILE_SIZE; ++i){
				pvalue += ds_A[ty][i] * ds_B[i][tx];
			}
		}
		__syncthreads();

		if(row < m && col < n){
			C[row * n + col] = pvalue;
		}
	}
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    const unsigned int BLOCK_SIZE = TILE_SIZE;
    dim3 dimGrid((n-1)/BLOCK_SIZE + 1, (m-1) / BLOCK_SIZE +1, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
 
    mysgemm <<< dimGrid, dimBlock >>> (m, n, k, A, B, C);	
}

__device__ __forceinline__ float sigmoid (float a){
	return 1.0 / (1.0 + exp (-a));
}

__global__ void sigmoid_kernel (const float *C, float *Csig, int len){
	int stride = gridDim.x * blockDim.x;
    	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	for(int i = tid; i < len; i += stride){
		Csig[i] = sigmoid (C[i]);
	}

}

void basicSigmoid(const float *C, float  *Csig, int len){
	dim3 dimBlock(256);
	int threadBlocks = (len + (dimBlock.x - 1)) / dimBlock.x;
	if (threadBlocks > 65520) threadBlocks = 65520;
	dim3 dimGrid(threadBlocks);

	sigmoid_kernel<<<dimGrid,dimBlock>>>(C, Csig, len);

}


__global__ void mysub(int m,int n, int k, const float *A, const float *B, float* C){
	for (int i = 0; i < (m * n); i++) {
		C[i] = A[i] - B[i];	
	}
}

void basicSub(int m,int n, int k, const float *A, const float *B, float* C){
    const unsigned int BLOCK_SIZE = TILE_SIZE;

    dim3 dimGrid((n-1)/BLOCK_SIZE + 1, (m-1) / BLOCK_SIZE + 1, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    mysub <<< dimGrid, dimBlock >>> (m,n, k, A, B, C);
}

